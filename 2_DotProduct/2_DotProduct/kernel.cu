#include "hip/hip_runtime.h"
/*
  Musa AYDIN, Fatih Sultan Mehmet Vakif University,
  Istanbul, Turkey
  maydin@fsm.edu.tr
  this program, calculates scaler dot products,
  using threads synchronization,
  first implementation in kernel each thread own private variable
  second, each thread share data using shared memory structure
*/
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <stdio.h>
#define N (10)
#define THREADS_PER_BLOCK 10

void init_matrix(int *a, int len) {
	int i;
	
	for ( i = 0; i < len; i++)
	{
		a[i] = i + 1;
	}
}
void print_array(int *a, int len) {
	int i;
	if (len > 16) {
		len = 16;
	}
	for (i = 0; i < len; i++)
	{
		printf("%d\n", a[i]);
	}
}

__global__ void dotSharedMem(int *a, int *b, int *c) {
	// temp de�i�keni her bir thread i�in private olan de�i�kendir,
	// her bir thread in kendi temp de�i�keni vard�r.
	__shared__ int temp[THREADS_PER_BLOCK];
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	 //temp[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x];
	 temp[id] = a[id] * b[id];	
	
	 __syncthreads();
	 if (0 == threadIdx.x) {
		 int sum = 0;
		 for (int i = 0; i < THREADS_PER_BLOCK; i++)
			 sum += temp[i];
		 atomicAdd(c, sum);
		// atomicAdd(&c[0], sum);
		 //*c = sum;
	 }

}
int main(void) {	

	int *a, *b, *c; // copies of a, b, c
	int *dev_a, *dev_b, *dev_c; // devices copies of a, b, c
	int size = N * sizeof(int); // allocate device copies of a, b, c

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, sizeof(int));
	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(sizeof(int));
	//*c = 0;

	init_matrix(a, N);
	print_array(a, N);
	init_matrix(b, N);
	
	// copy inputs to device
	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
	dotSharedMem << < N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(dev_a, dev_b, dev_c);

	// copy device result back to host copy of c
	hipMemcpy(c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

	printf("dot product result = %d\n", c[0]);
	//printf("%d\n", c);
	//print_array(c, N);

	free(a); free(b); free(c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}


